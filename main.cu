#include "hip/hip_runtime.h"
/*
   Kernel koji množi dvije kvadratne matrice. Matrice su dinamički alocirane i
   zadane su po recima. 
    C = A B
    Matrice su kvadratne dimenzije,  N x N. 
    Verzija s optimizacijom dohvata iz memorije (blok algoritam)
    i dimenzijom matrice koja ne mora biti djeljiva s dimenzijom bloka. 
*/


__global__
void matMulKernel(float * A, float * B, float * C, int N)
{
    // VAŠ KOD
}

void error_h(hipError_t error, const char * file, int line){
     if(error != hipSuccess){
         std::cerr << hipGetErrorString(error) << " at file " << file << " in line " << line << "\n";
         std::exit(EXIT_FAILURE);
     }
}

__host__
bool checkResult(float * A, float * B, float * C, int N){
    float EPS = 1E-6f;
    bool res = true;
    float max_error = 0.0f;
    float max_value = 0.0f;
    for(int i=0; i<N; ++i)
    {
        for(int j=0; j<N; ++j)
        {
	       float tmp = 0.0;
           for(int k=0; k<N; ++k)
		       tmp += A[i*N+k] * B[k*N+j];

	       float diff = fabs(tmp - C[i*N+j]);
	       float val  = fabs(tmp);
	       if(diff > max_error) max_error = diff;
	       if(val  > max_value) max_value = val;  
	    } // for po j
    } // for po i

    
    if(max_error > EPS*max_value){
        std::cout << "(Max error = " << max_error << ", max value = " << max_value 
                  << ", relative error = " <<  max_error/max_value << ")";
        res = false;
    }
    return res;
}


int main()
{
    // VAŠ KOD   
    return 0;
}

